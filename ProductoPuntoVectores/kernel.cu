﻿/**
 * Laboratorio 6
 * 
 * Programacion de microprocesadores
 * 
 * Producto punto entre dos vectores de 768 elementos
 * 
 * 
 * Angel Castellanos 21700
 * Jose Pablo Santisteban 21153
 */

#include <stdio.h>

  
#include <hip/hip_runtime.h>


__global__ void
dotProduct(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        //multiplicacion de elementos de vectores
        C[i] = A[i] * B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    //Variable que contendra el resultado de la operacion
    float resultado = 0;

    // codigo de error a manejar
    hipError_t err = hipSuccess;

    // imprime el tamaño de los vectores
    int numElements = 768;
    size_t size = numElements * sizeof(float);
    printf("[Suma de vectores de %d elementos]\n", numElements);

    // Reserva de memoria para los vectores en el host 
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // verifica que la reserva de memoria haya sido exitosa
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Error al asignar la memoria!\n");
        exit(EXIT_FAILURE);
    }

    // Inicializa los vectores en el host 
    for (int i = 0; i < numElements; ++i)
    {
        float numRand1 = (500 + rand() % (5000 - 500)) / (float)RAND_MAX;
        float numRand2 = (500 + rand() % (5000 - 500)) / (float)RAND_MAX;
        h_A[i] = numRand1;
        h_B[i] = numRand2;
    }

    // Reserva de memoria para los vectores en el device
    float* d_A = NULL;
    err = hipMalloc((void**)&d_A, size);

    // verifica que la reserva de memoria haya sido exitosa en el device 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo en reserval la memoria, ver código %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reserva de memoria para los vectores en el device y verifica que la reserva de memoria haya sido exitosa
    float* d_B = NULL;
    err = hipMalloc((void**)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo en reservar la memoria en el device del vector B, ver código %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo en reservar la memoria en el device del vector C, ver código %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copia los vectores A y B del host al device
    printf("Copiando los vectores del host al device...\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // verifica que la copia de memoria de los vectores A y B haya sido exitosa
    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al copiar el vector A del host al device (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al copiar el vector B del host al device (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("Lanzamiento del kernel CUda con  %d bloques de %d hilos\n", blocksPerGrid, threadsPerBlock);
    dotProduct <<<blocksPerGrid, threadsPerBlock >>> (d_A, d_B, d_C, numElements);
    err = hipGetLastError();


    // verifica que el lanzamiento del kernel haya sido exitoso
    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al multiplicar los vectores (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copia el resultado del device (d_C) al host (h_C)
    printf("Copiando el resultado del device al host...\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al copiar el vector c del host al devide (código de error  %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //prueba que la suma de los vectores sea correcta
    for (int i = 0; i < numElements; ++i)
    {
        float multiplicacion = h_A[i] * h_B[i];
        if (fabs(multiplicacion / h_C[i]) != 1)
        {
            fprintf(stderr, "Resultado de verificación falló en el elemento %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Prueba pasada\n");


    //Suma el resultado de la suma de los vectores
    for (int i = 0; i < numElements; i++)
    {
        resultado = resultado + (float)h_C[i];
    }

    printf("El resultado total es: %f \n", resultado);

    

    // Libera la memoria reservada en el device
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al liberar el vector A del device (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al liberar el vector B del device (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo al liberar el vector C del device (código de error %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Libera la memoria reservada en el host
    free(h_A);
    free(h_B);
    free(h_C);


    // Reinicia el device y verifica que la reinicialización haya sido exitosa
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Fallo a desinicializar el device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Finalizado\n");
    return 0;
}

